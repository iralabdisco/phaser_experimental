#include "hip/hip_runtime.h"
#include <complex.h>
#include <hipfft/hipfft.h>
#include <gflags/gflags.h>
#include <glog/logging.h>
#include <vector>

#include "phaser/backend/correlation/spatial-correlation-cuda.h"

DEFINE_double(phase_gpu_batch, 1, "");

namespace phaser_core {

SpatialCorrelationCuda::SpatialCorrelationCuda(const uint32_t voxels_per_dim)
    : n_voxels_total_(voxels_per_dim * voxels_per_dim * voxels_per_dim),
      n_voxels_per_dim_(voxels_per_dim) {
  // Allocate memory for the FFT and IFFT.
  const uint32_t data_size = sizeof(hipfftDoubleComplex) * n_voxels_total_;
  hipMalloc(reinterpret_cast<void**>(&F_), data_size);
  hipMalloc(reinterpret_cast<void**>(&G_), data_size);
  hipMalloc(reinterpret_cast<void**>(&C_), data_size);
  c_ = new double[n_voxels_total_];

  // Create the cuda plans for two FFTs and one IFFT.
  hipfftPlan3d(
      &f_plan_, voxels_per_dim, voxels_per_dim, voxels_per_dim, HIPFFT_D2Z);
  hipfftPlan3d(
      &c_plan_, voxels_per_dim, voxels_per_dim, voxels_per_dim, HIPFFT_Z2D);
}

SpatialCorrelationCuda::~SpatialCorrelationCuda() {
  hipFree(F_);
  hipFree(G_);
  hipFree(C_);
  hipfftDestroy(f_plan_);
  hipfftDestroy(c_plan_);
  delete[] c_;
}

// CUDA kernel for the spatial phase correlation.
// Simple complex multiplication version.
__global__ void correlation(
    hipfftDoubleComplex* F, hipfftDoubleComplex* G, hipfftDoubleComplex* C,
    uint32_t size) {
  // unique block index inside a 3D block grid
  const uint64_t blockId = blockIdx.x                             // 1D
                           + blockIdx.y * gridDim.x               // 2D
                           + gridDim.x * gridDim.y * blockIdx.z;  // 3D

  // global unique thread index, block dimension uses only x-coordinate
  const uint64_t idx = blockId * blockDim.x + threadIdx.x;

  C[idx].x = F[idx].x * G[idx].x - F[idx].y * (-G[idx].y);
  C[idx].y = F[idx].x * (-G[idx].y) + F[idx].y * G[idx].x;
}

double* SpatialCorrelationCuda::correlateSignals(
    const std::vector<Eigen::VectorXd*>& f,
    const std::vector<Eigen::VectorXd*>& g) {
  CHECK_GT(f.size(), 0u);
  CHECK_GT(g.size(), 0u);
  // Perform the two FFTs on the discretized signals.
  VLOG(1) << "Performing FFT on the first point cloud.";
  double* d_input;
  hipMalloc(
      reinterpret_cast<void**>(&d_input), sizeof(double) * n_voxels_total_);
  hipMemcpy(
      d_input, f[0]->data(), sizeof(double) * n_voxels_total_,
      hipMemcpyHostToDevice);
  // R2C is for floats, D2Z for doubles.
  if (hipfftExecD2Z(f_plan_, d_input, F_) != HIPFFT_SUCCESS) {
    LOG(FATAL) << "Forward cufft transform failed.";
  }

  VLOG(1) << "Performing FFT on the second point cloud.";
  hipMemcpy(
      d_input, g[0]->data(), sizeof(double) * n_voxels_total_,
      hipMemcpyHostToDevice);
  hipfftExecD2Z(f_plan_, d_input, G_);

  // Correlate the signals in the frequency domain.
  const auto tile_width = 32;
  const auto n_threads =
      std::ceil(static_cast<float>(n_voxels_per_dim_) / tile_width);
  dim3 dimBlock(n_threads, n_threads, n_threads);
  dim3 dimGrid(tile_width, tile_width, 1);
  correlation<<<dimGrid, dimBlock>>>(F_, G_, C_, n_voxels_total_);

  // Perform the IFFT on the correlation tensor.
  VLOG(1) << "Performing IFFT on correlation.";
  hipfftExecZ2D(c_plan_, C_, d_input);
  hipMemcpy(
      c_, d_input, sizeof(double) * n_voxels_total_, hipMemcpyDeviceToHost);
  return c_;
}

}  // namespace phaser_core
